#include "composite.h"
#include "util/error.h"
#include <iostream>

int CompositeLens::destroy() {
    if (m_cuda) {
        gpuErrchk(hipFree(m_lenses));
    } else {
        free(m_lenses);
    }
    return 0;
}

CompositeLens CompositeLensBuilder::getCuLens() {
    Plummer *lens_ptr;
#ifdef __HIPCC__
    size_t size = m_lenses.size();
    if (size == 0) {
        std::cerr << "No lenses added" << std::endl;
        throw(-1);
    }
    size_t numbytes = sizeof(Plummer) * size;
    gpuErrchk(hipMalloc(&lens_ptr, numbytes));
    gpuErrchk(
        hipMemcpy(lens_ptr, &m_lenses[0], numbytes, hipMemcpyHostToDevice));
#else
    CompositeLens *lens_ptr = nullptr;
#endif
    return CompositeLens(lens_ptr, size, true);
}
