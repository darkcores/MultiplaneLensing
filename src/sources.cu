#include "sources.h"

#include "util/error.h"

#include <iostream>

SourcePlane SourcePlaneBuilder::getCuPlane() {
    cuda = true;
#ifdef __HIPCC__
    // dev_m_points = m_points;
    // ptr = thrust::raw_pointer_cast(&dev_m_points[0]);
    size_t size = sizeof(SourceData) * m_points.size();
	/*
    if (size == 0) {
        std::cerr << "No sources added" << std::endl;
        std::terminate();
    }
	*/
    gpuErrchk(hipMalloc(&ptr, size));
    gpuErrchk(hipMemcpy(ptr, &m_points[0], size, hipMemcpyHostToDevice));
#else
    ptr = nullptr;
#endif
    return SourcePlane(m_redshift, ptr, m_points.size(), true);
}

int SourcePlane::destroy() {
    if (m_points_ptr) {
        if (m_cuda) {
#ifdef __HIPCC__
            gpuErrchk(hipFree(m_points_ptr));
#endif
        } else {
            free(m_points_ptr);
        }
        m_points_ptr = NULL;
    }
    return 0;
}