#include "multiplane_cu.h"

#include "multiplane.h"
#include "util/error.h"

MultiPlaneContext::MultiPlaneContext(const double angularUnit,
                                     const Cosmology cosmology)
    : m_angularUnit(angularUnit), m_cosmology(cosmology) {
    m_theta_x = nullptr;
    m_theta_y = nullptr;
    m_beta_x = nullptr;
    m_beta_y = nullptr;
    m_multiplane = nullptr;
}

MultiPlaneContext::~MultiPlaneContext() {
    if (m_theta_x)
        gpuErrchk(hipFree(m_theta_x));
    if (m_theta_y)
        gpuErrchk(hipFree(m_theta_y));
    if (m_beta_x)
        gpuErrchk(hipFree(m_beta_x));
    if (m_beta_y)
        gpuErrchk(hipFree(m_beta_y));
    if (m_multiplane)
        gpuErrchk(hipFree(m_multiplane));
}

CompositeLensBuilder
MultiPlaneContext::buildLens(const double Dd,
                             const std::vector<PlummerParams> &params) {
	CompositeLensBuilder builder;
	for (auto &param : params) {
		auto position = param.position * m_angularUnit;
		Plummer plum(Dd, param.mass, param.angularwidth);
		builder.addLens(plum, position);
	}
	return builder;
}

int MultiPlaneContext::init(
    const std::vector<float> &lensRedshifts,
    const std::vector<std::vector<PlummerParams>> &params,
    const std::vector<float> &sourceRedshifts) {
    MultiplaneBuilder planebuilder(m_cosmology);

	// Setup lenses
    for (size_t i = 0; i < lensRedshifts.size(); i++) {
        double Dd = m_cosmology.angularDiameterDistance(lensRedshifts[i]);
        auto lens = buildLens(Dd, params[i]);
		lens.setRedshift(lensRedshifts[i]);
        planebuilder.addPlane(lens);
    }

	// Setup sources
	for (auto z : sourceRedshifts) {
		// TODO, maybe just add a function that takes this vector
		SourcePlaneBuilder sp(z);
		auto plane = sp.getCuPlane();
		planebuilder.addSourcePlane(plane);
	}

	// Build multiplane

    return 0;
}
