#include "hip/hip_runtime.h"
#include "multiplane.h"

#include "util/error.h"
#include <algorithm>
#include <iostream>
#include <thrust/device_vector.h>

Multiplane MultiplaneBuilder::getCuMultiPlane() {
    std::vector<CompositeLens> data;
    std::vector<int> subl_size;

    for (size_t i = 0; i < m_builders.size(); i++) {
        subl_size.push_back(m_builders[i].length());
        auto lens = m_builders[i].getCuLens();
        data.push_back(lens);
    }

    if (data.size() == 0 || m_source_z.size() == 0) {
        std::cerr << "No lens and/or source planes given " << data.size() << "-"
                  << m_source_z.size() << std::endl;
        throw(-1);
    }

    prepare();

    CompositeLens *lens_ptr;
    float *src_ptr, *dist_lens_ptr, *dist_src_ptr;

    size_t lens_size = sizeof(CompositeLens) * data.size();
    gpuErrchk(hipMalloc(&lens_ptr, lens_size));
    gpuErrchk(
        hipMemcpy(lens_ptr, &data[0], lens_size, hipMemcpyHostToDevice));

    size_t src_size = sizeof(float) * m_source_z.size();
    gpuErrchk(hipMalloc(&src_ptr, src_size));
    gpuErrchk(
        hipMemcpy(src_ptr, &m_source_z[0], src_size, hipMemcpyHostToDevice));

    size_t dist_lens_size = sizeof(float) * m_dists_lenses.size();
    gpuErrchk(hipMalloc(&dist_lens_ptr, dist_lens_size));
    gpuErrchk(hipMemcpy(dist_lens_ptr, &m_dists_lenses[0], dist_lens_size,
                         hipMemcpyHostToDevice));

    size_t dist_src_size = sizeof(float) * m_dists_sources.size();
    gpuErrchk(hipMalloc(&dist_src_ptr, dist_src_size));
    gpuErrchk(hipMemcpy(dist_src_ptr, &m_dists_sources[0], dist_src_size,
                         hipMemcpyHostToDevice));

    return Multiplane(lens_ptr, data.size(), src_ptr, m_source_z.size(),
                      dist_lens_ptr, dist_src_ptr, m_dist_offsets, subl_size,
                      true);
}

Multiplane *MultiplaneBuilder::getCuMultiPlanePtr() {
    std::vector<CompositeLens> data;
    std::vector<int> subl_size;

    for (size_t i = 0; i < m_builders.size(); i++) {
        subl_size.push_back(m_builders[i].length());
        auto lens = m_builders[i].getCuLens();
        data.push_back(lens);
    }

    if (data.size() == 0 || m_source_z.size() == 0) {
        std::cerr << "No lens and/or source planes given " << data.size() << "-"
                  << m_source_z.size() << std::endl;
        throw(-1);
    }

    prepare();

    CompositeLens *lens_ptr;
    float *src_ptr, *dist_lens_ptr, *dist_src_ptr;

    size_t lens_size = sizeof(CompositeLens) * data.size();
    gpuErrchk(hipMalloc(&lens_ptr, lens_size));
    gpuErrchk(
        hipMemcpy(lens_ptr, &data[0], lens_size, hipMemcpyHostToDevice));

    size_t src_size = sizeof(float) * m_source_z.size();
    gpuErrchk(hipMalloc(&src_ptr, src_size));
    gpuErrchk(
        hipMemcpy(src_ptr, &m_source_z[0], src_size, hipMemcpyHostToDevice));

    size_t dist_lens_size = sizeof(float) * m_dists_lenses.size();
    gpuErrchk(hipMalloc(&dist_lens_ptr, dist_lens_size));
    gpuErrchk(hipMemcpy(dist_lens_ptr, &m_dists_lenses[0], dist_lens_size,
                         hipMemcpyHostToDevice));

    size_t dist_src_size = sizeof(float) * m_dists_sources.size();
    gpuErrchk(hipMalloc(&dist_src_ptr, dist_src_size));
    gpuErrchk(hipMemcpy(dist_src_ptr, &m_dists_sources[0], dist_src_size,
                         hipMemcpyHostToDevice));

    return new Multiplane(lens_ptr, data.size(), src_ptr, m_source_z.size(),
                          dist_lens_ptr, dist_src_ptr, m_dist_offsets,
                          subl_size, true);
}

int Multiplane::destroy() {
    // Destroy children
    if (m_cuda) {
        // So we copy them back to cpu first and then destroy TODO:
        // consider just keeping a vector of them in memory for
        // cleanliness
        size_t psize = m_lenses_size * sizeof(CompositeLens);
        CompositeLens *pptr = (CompositeLens *)malloc(psize);
        cpuErrchk(pptr);
        gpuErrchk(hipMemcpy(pptr, m_lenses, psize, hipMemcpyDeviceToHost));
        for (int i = 0; i < m_lenses_size; i++) {
            pptr[i].destroy();
        }
        free(pptr);
    } else {
        for (int i = 0; i < m_lenses_size; i++) {
            m_lenses[i].destroy();
        }
    }

    // Free memory
    if (m_cuda) {
        gpuErrchk(hipFree(m_lenses));
        gpuErrchk(hipFree(m_sources));
        gpuErrchk(hipFree(m_dist_lenses));
        gpuErrchk(hipFree(m_dist_sources));
        if (m_alphas)
            gpuErrchk(hipFree(m_alphas));
    } else {
        free(m_lenses);
        free(m_sources);
        free(m_dist_lenses);
        free(m_dist_sources);
    }
    m_lenses = nullptr;
    m_sources = nullptr;
    m_dist_lenses = nullptr;
    m_dist_sources = nullptr;
    m_alphas = nullptr;
    return 0;
}

__global__ void mp_traceThetaGlobal(const long n,
                                    const float2 *__restrict__ thetas,
                                    float2 *__restrict__ betas,
                                    const float *__restrict__ dist_lenses,
                                    const float *__restrict__ dist_sources,
                                    const int numlenses, const int offset,
                                    const CompositeLens *__restrict__ lenses) {
    const long z = min((long)blockIdx.x * blockDim.x + threadIdx.x, n - 1);

    const int MAX_PLANES = 128;
    float2 alphas[MAX_PLANES];

    float2 last_theta;
    int l = 0;
    for (int i = 0; i <= numlenses; i++) {
        auto t = thetas[z];
        if (i > 0) {
            const int idx = i - 1;
            alphas[idx] = lenses[idx].getAlpha(last_theta);
        }
        for (int j = 0; j < i; j++) {
            const float2 a = alphas[j];
            const float dist = dist_lenses[l];
            t.x -= a.x * dist;
            t.y -= a.y * dist;
            l++;
        }
        last_theta = t;
    }

    l = offset;
    auto t = thetas[z];
    for (int i = 0; i < numlenses; i++) {
        const float2 a = alphas[i];
        const float dist = dist_sources[l];
        t.x -= a.x * dist;
        t.y -= a.y * dist;
        l++;
    }
    betas[z] = t;
}

int Multiplane::traceThetas(const float2 *thetas, float2 *betas, const size_t n,
                            const int plane) const {
    size_t offset = 0;
    for (int i = 0; i < plane; i++) {
        int s = m_dist_offsets[i];
        offset += s;
    }

    int numlenses = m_dist_offsets[plane];
    mp_traceThetaGlobal<<<(n / 128) + 1, 128>>>(n, thetas, betas, m_dist_lenses,
                                                m_dist_sources, numlenses,
                                                offset, m_lenses);
    gpuErrchk(hipGetLastError());

    return 0;
}

__global__ void mp_updateMasses(const size_t n, const float *__restrict__ masses,
                                const int lens,
                                CompositeLens *__restrict__ lenses) {
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        // printf("Mass %i: %f\n", i, masses[i]);
        lenses[lens].update(masses[i], i);
    }
}

void Multiplane::updateMassesCu(const std::vector<std::vector<float>> &masses) {
    thrust::device_vector<float> mass;
    float *ptr;
    for (size_t i = 0; i < masses.size(); i++) {
        mass = masses[i];
        ptr = thrust::raw_pointer_cast(&mass[0]);
        size_t size = masses[i].size();
        mp_updateMasses<<<(size / 64) + 1, 64>>>(size, ptr, i, m_lenses);
    }
}
