#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

#include <composite.h>
#include <thrust/device_vector.h>
#include <util/constants.h>
#include <util/cosmology.h>

CompositeLensBuilder createGrid(double Dd, int N, double width, double height,
                                double angularwidth, double mass) {
    CompositeLensBuilder lensbuilder(Dd); // Redshift not used in these tests.
    double xstart = -width / 2;
    double xend = width / 2;
    double xstep = width / (N - 1);
    double ystart = -height / 2;
    double yend = height / 2;
    double ystep = height / (N - 1);
    // This works for this test, but might not always work TODO
    for (double x = xstart; x <= xend; x += xstep) {
        for (double y = ystart; y <= yend; y += ystep) {
            Plummer plum(Dd, mass, angularwidth, 1 / ANGLE_ARCSEC,
                         float2{.x = (float)x, .y = (float)y});
            lensbuilder.addLens(plum);
        }
    }
    return lensbuilder;
}

__global__ void alphaCompCalc(int n, float2 *thetas, float2 *alphas,
                              CompositeLens lens) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        alphas[i] = lens.getAlpha(thetas[i]);
    }
}

TEST(CompositeTests, TestAlpha) {
    Cosmology cosm(0.7, 0.3, 0.0, 0.7);
    double z_d = 0.4;
    auto Dd = cosm.angularDiameterDistance(z_d);
    auto lensbuilder = createGrid(Dd, 3, 15, 15, 5, 1e13 * MASS_SOLAR);
    auto lens = lensbuilder.getCuLens();
    float2 point{.x = 1.0, .y = 2.0};
    thrust::device_vector<float2> d_thetas(1), d_alphas(1);
    d_thetas[0] = point;
    float2 *thetaptr = thrust::raw_pointer_cast(&d_thetas[0]);
    float2 *alphaptr = thrust::raw_pointer_cast(&d_alphas[0]);

    alphaCompCalc<<<1, 32>>>(1, thetaptr, alphaptr, lens);

    float2 alpha = d_alphas[0];
    // auto alpha = lens.getAlpha(point);
    // alpha *= ANGLE_ARCSEC;
    // EXPECT_EQ(alpha.x(), 2.01259882e-05);
    // EXPECT_EQ(alpha.y(), 3.91372304e-05);
    // EXPECT_LT(abs(alpha.x() - 2.01259882e-05), 1e-10);
    // EXPECT_LT(abs(alpha.y() - 3.91372304e-05), 1e-10);
    lens.destroy();
}
